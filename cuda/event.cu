#include <hip/hip_runtime.h>
#include <iostream>

// 简单的CUDA内核函数
__global__ void simpleKernel(int* data, int value) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    data[idx] = value;
}

int main() {
    const int dataSize = 256;
    const int bytes = dataSize * sizeof(int);

    // 分配主机和设备内存
    int* h_data = (int*)malloc(bytes);
    int* d_data;
    hipMalloc(&d_data, bytes);

    // 创建CUDA事件
    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);

    // 记录开始事件
    hipEventRecord(startEvent, 0);

    // 启动CUDA内核
    simpleKernel<<<dataSize / 256, 256>>>(d_data, 42);

    // 记录结束事件
    hipEventRecord(stopEvent, 0);

    // 等待事件完成
    hipEventSynchronize(stopEvent);

    // 计算内核执行时间
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, startEvent, stopEvent);

    std::cout << "Kernel execution time: " << milliseconds << " ms" << std::endl;

    // 释放资源
    hipEventDestroy(startEvent);
    hipEventDestroy(stopEvent);
    hipFree(d_data);
    free(h_data);

    return 0;
}
