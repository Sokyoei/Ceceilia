#include <iostream>

#include <hip/hip_runtime.h>

#include "Ahri.cuh"

int main(int argc, char const* argv[]) {
    int device_count;
    CUDA_CHECK(hipGetDeviceCount(&device_count));
    std::cout << "CUDA Device: " << device_count << '\n';

    if (device_count > 0) {
        int device_id = 0;
        hipDeviceProp_t device_prop;
        CUDA_CHECK(hipGetDeviceProperties(&device_prop, device_id));
        std::cout << device_prop.name << '\n';

        int attr;
        CUDA_CHECK(hipDeviceGetAttribute(&attr, hipDeviceAttributeL2CacheSize, device_id));
        std::cout << "L2 cache size: " << attr / 1024 << " KB" << std::endl;
    }

    return 0;
}
