#include <iostream>

#include <hip/hip_runtime.h>
#include <fmt/core.h>

#include "Ahri/Ahri.cuh"

int main(int argc, char const* argv[]) {
    int device_count;
    CUDA_CHECK(hipGetDeviceCount(&device_count));
    fmt::println("Find CUDA Device: {}", device_count);

    if (device_count > 0) {
        int device_id = 0;
        hipDeviceProp_t device_prop;
        CUDA_CHECK(hipGetDeviceProperties(&device_prop, device_id));
        fmt::println("NVIDIA GPU name: {}", device_prop.name);

        int attr;
        CUDA_CHECK(hipDeviceGetAttribute(&attr, hipDeviceAttributeL2CacheSize, device_id));
        fmt::println("L2 cache size: {}KB", attr / 1024);
    }

    return 0;
}
